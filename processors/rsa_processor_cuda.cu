#include "hip/hip_runtime.h"
#include <omp.h>
#include <iostream>
#include <sstream>
#include <vector>
#include <hip/hip_runtime.h>
#include "../includes/rsa_processor_cuda.h"

// CUDA error checking macro
#define CUDA_CHECK(call) \
do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA Error in " << __FILE__ << " at line " << __LINE__ << ": " \
                  << hipGetErrorString(err) << std::endl; \
        throw std::runtime_error("CUDA Error: " + std::string(hipGetErrorString(err))); \
    } \
} while(0)

__device__ int mod_exp_gpu(unsigned long long base, unsigned long long exp, unsigned long long mod) {
    unsigned long long result = 1;
    base = base % mod; // Handle large base values
    while (exp > 0) {
        if (exp % 2 == 1) {
            result = (result * base) % mod;
        }
        base = (base * base) % mod;
        exp /= 2;
    }

    return (int)result;
}

__device__ int encrypt_gpu(struct Key* key, int message) {
    return mod_exp_gpu(message, key->e, key->public_key);
}

__device__ int decrypt_gpu(struct Key* key, int cipher) {
    return mod_exp_gpu(cipher, key->private_key, key->public_key);
}

// CUDA kernel for encryption
__global__ void encrypt_kernel(struct Key* key, char* message, int* cipher_list, int message_length) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < message_length) {
        int target = message[idx];
        cipher_list[idx] = encrypt_gpu(key, target);
    }
}

// CUDA kernel for decryption
__global__ void decrypt_kernel(struct Key* key, int* cipher_list, char* decrypted_chars, int cipher_length) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < cipher_length) {
        int cipher = cipher_list[idx];
        decrypted_chars[idx] = (char)decrypt_gpu(key, cipher);
    }
}

// Helper function to copy the key to device memory
void copy_key_to_device(struct Key* key, struct Key** d_key) {
    CUDA_CHECK(hipMalloc((void**)d_key, sizeof(struct Key)));
    CUDA_CHECK(hipMemcpy(*d_key, key, sizeof(struct Key), hipMemcpyHostToDevice));
}

// Encryption function with CUDA and error checking
std::string encrypt_string_gpu(struct Key* key, std::string message, int block_size) {
    try {
        int message_length = message.length();
        
        // Allocate device memory
        char* d_message = nullptr;
        int* d_cipher_list = nullptr;
        struct Key* d_key = nullptr; // Device pointer for key
        
        CUDA_CHECK(hipMalloc(&d_message, message_length * sizeof(char)));
        CUDA_CHECK(hipMalloc(&d_cipher_list, message_length * sizeof(int)));
        
        // Copy host data to device
        CUDA_CHECK(hipMemcpy(d_message, message.c_str(), message_length * sizeof(char), hipMemcpyHostToDevice));
        
        // Copy key to device
        copy_key_to_device(key, &d_key);
        
        // Configure grid and block dimensions
        int grid_size = (message_length + block_size - 1) / block_size;
        
        // Launch encryption kernel
        encrypt_kernel<<<grid_size, block_size>>>(d_key, d_message, d_cipher_list, message_length);
        
        // Check for kernel launch errors
        CUDA_CHECK(hipGetLastError());
        
        // Synchronize to ensure kernel completion
        CUDA_CHECK(hipDeviceSynchronize());
        
        // Allocate host memory for results
        std::vector<int> cipher_list(message_length);
        
        // Copy results back to host
        CUDA_CHECK(hipMemcpy(cipher_list.data(), d_cipher_list, message_length * sizeof(int), hipMemcpyDeviceToHost));
        
        // Free device memory
        CUDA_CHECK(hipFree(d_message));
        CUDA_CHECK(hipFree(d_cipher_list));
        CUDA_CHECK(hipFree(d_key));
        
        // Convert to string
        std::stringstream cipher_text_stream;
        for(int cipher : cipher_list) {
            cipher_text_stream << cipher << " ";
        }
        
        return cipher_text_stream.str();
    }
    catch (const std::exception& e) {
        std::cerr << "Error in encrypt_string: " << e.what() << std::endl;
        throw;
    }
}

// Decryption function with CUDA and error checking
std::string decrypt_string_gpu(struct Key* key, std::string cipher_text, int block_size) {
    try {
        // Parse cipher text
        std::istringstream iss(cipher_text);
        std::vector<int> cipher_list;
        int temp;
        while(iss >> temp) {
            cipher_list.push_back(temp);
        }
        int cipher_length = cipher_list.size();
        
        // Allocate device memory
        int* d_cipher_list = nullptr;
        char* d_decrypted_chars = nullptr;
        struct Key* d_key = nullptr; // Device pointer for key
        
        CUDA_CHECK(hipMalloc(&d_cipher_list, cipher_length * sizeof(int)));
        CUDA_CHECK(hipMalloc(&d_decrypted_chars, cipher_length * sizeof(char)));
        
        // Copy input data to device
        CUDA_CHECK(hipMemcpy(d_cipher_list, cipher_list.data(), cipher_length * sizeof(int), hipMemcpyHostToDevice));
        
        // Copy key to device
        copy_key_to_device(key, &d_key);
        
        // Configure grid and block dimensions
        int grid_size = (cipher_length + block_size - 1) / block_size;
        
        // Launch decryption kernel
        decrypt_kernel<<<grid_size, block_size>>>(d_key, d_cipher_list, d_decrypted_chars, cipher_length);
        
        // Check for kernel launch errors
        CUDA_CHECK(hipGetLastError());
        
        // Synchronize to ensure kernel completion
        CUDA_CHECK(hipDeviceSynchronize());
        
        // Allocate host memory for results
        std::vector<char> decrypted_chars(cipher_length);
        
        // Copy results back to host
        CUDA_CHECK(hipMemcpy(decrypted_chars.data(), d_decrypted_chars, cipher_length * sizeof(char), hipMemcpyDeviceToHost));
        
        // Free device memory
        CUDA_CHECK(hipFree(d_cipher_list));
        CUDA_CHECK(hipFree(d_decrypted_chars));
        CUDA_CHECK(hipFree(d_key));
        
        return std::string(decrypted_chars.begin(), decrypted_chars.end());
    }
    catch (const std::exception& e) {
        std::cerr << "Error in decrypt_string: " << e.what() << std::endl;
        throw;
    }
}

// Optional: Device information function
void printDeviceInfo() {
    int deviceCount;
    CUDA_CHECK(hipGetDeviceCount(&deviceCount));
    
    for (int dev = 0; dev < deviceCount; dev++) {
        hipDeviceProp_t deviceProp;
        CUDA_CHECK(hipGetDeviceProperties(&deviceProp, dev));
        
        std::cout << "Device " << dev << ": " << deviceProp.name << std::endl;
        std::cout << "  Compute Capability: " << deviceProp.major << "." << deviceProp.minor << std::endl;
        std::cout << "  Total Global Memory: " << deviceProp.totalGlobalMem / (1024 * 1024) << " MB" << std::endl;
        std::cout << "  Multiprocessor Count: " << deviceProp.multiProcessorCount << std::endl;
    }
}

