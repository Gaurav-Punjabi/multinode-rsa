// gpu_properties.cpp
#include <iostream>
#include <hip/hip_runtime.h>

void printDeviceProperties() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    
    if (deviceCount == 0) {
        std::cout << "No CUDA devices found." << std::endl;
        return;
    }

    for (int dev = 0; dev < deviceCount; dev++) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
        
        std::cout << "Device " << dev << ": " << deviceProp.name << std::endl;
        std::cout << "  Max Threads per Block: " << deviceProp.maxThreadsPerBlock << std::endl;
        std::cout << "  Max Threads per Multiprocessor: " << deviceProp.maxThreadsPerMultiProcessor << std::endl;
        std::cout << "  Max Blocks per MultiProcessor: " << deviceProp.maxBlocksPerMultiProcessor << std::endl;
        std::cout << "  Max Shared Memory per Block: " << deviceProp.sharedMemPerBlock / (1024) << " KB" << std::endl;
        std::cout << "  Total Global Memory: " << deviceProp.totalGlobalMem / (1024 * 1024) << " MB" << std::endl;
        std::cout << "  Compute Capability: " << deviceProp.major << "." << deviceProp.minor << std::endl;
        std::cout << "  Multi-Processor Count: " << deviceProp.multiProcessorCount << std::endl;
    }
}

int main() {
    printDeviceProperties();
    return 0;
}

